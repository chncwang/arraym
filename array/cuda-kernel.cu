#include "hip/hip_runtime.h"
#include "cuda-utils.h"

#ifdef WITH_CUDA

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "cuda-kernel.cuh"
#include <thrust/device_vector.h>
#include <thrust/fill.h>

DECLARE_NAMESPACE_NLL

namespace cuda
{
   template<typename T>
   __global__ void _kernel_init(T* ptr, const size_t nb_elements, const T val)
   {
      int tidx = threadIdx.x + blockDim.x * blockIdx.x;
      const int stride = blockDim.x * gridDim.x;
      for (; tidx < nb_elements; tidx += stride)
      {
         ptr[tidx] = val;
      }

      //
      // hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize,
      // &_kernel_init<T>, 0, 0);
      //
   }

   /*
   template<typename T>
   void kernel_init(T* ptr, const T val, const size_t nb_elements)
   {      
      // TODO see https://devblogs.nvidia.com/parallelforall/cuda-pro-tip-occupancy-api-simplifies-launch-configuration/
      // hipOccupancyMaxPotentialBlockSize
      const size_t block_size = 256;
      _kernel_init<T> <<<(nb_elements + block_size - 1 ) / block_size, block_size >>>(ptr, val, nb_elements);
      hipDeviceSynchronize();
      cudaCheck();
      
      //thrust::device_ptr<T> dev_ptr(ptr);
      //thrust::fill(dev_ptr, dev_ptr + nb_elements, val);
   }*/

   template<typename T>
   void kernel_init(T* ptr, const size_t nb_elements, const T val)
   {
      thrust::device_ptr<T> dev_ptr(ptr);
      thrust::fill(dev_ptr, dev_ptr + nb_elements, val);
   }

   template<typename T>
   void kernel_copy(const T* input, const size_t nb_elements, T* output)
   {
      thrust::device_ptr<T> dev_ptr_in(const_cast<T*>(input));
      thrust::device_ptr<T> dev_ptr_out(output);
      thrust::copy(dev_ptr_in, dev_ptr_in + nb_elements, dev_ptr_out);
   }

   template ARRAY_API void kernel_init(float* ptr, const size_t nb_elements, const float val);
   template ARRAY_API void kernel_copy(const float* input, const size_t nb_elements, float* output);

}

DECLARE_NAMESPACE_NLL_END

#endif